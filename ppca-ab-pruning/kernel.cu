#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

// added by me
#include <iostream>
#include <queue>
#include <map>

#include "board.h"
#include "minmax.h"

__global__ void minmaxKernel(int taskCount, long long* results, int* data, unsigned int dataSize) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    /*for (; i < count; i += blockDim.x * gridDim.x)
        c[i] = (1 << i) + 1;*/
}

Board FindBestMove(Board& board, GAME_CHAR player, int depth) {
    auto cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        cerr << "hipSetDevice failed! Do you have a CUDA-capable GPU installed?";
        exit(1);
    }

    int N = 2 << 13;
    ppca::minmax mmAlg;

    int searchedDepth;

    int blockSize;      // The launch configurator returned block size 
    int minGridSize;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch 
    int gridSize;       // The actual grid size needed, based on input size

    auto tasks = mmAlg.GetTasks(board, player, N, depth, searchedDepth);

    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, minmaxKernel, 0, tasks.size());

    // Round up according to array size 
    gridSize = (tasks.size() + blockSize - 1) / blockSize;

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Occupancy calculator elapsed time:  %3.3f ms \n", time);

    int *dev_data, *host_data;
    long long *dev_results, *host_results;
    auto size = mmAlg.ConvertToGpuData(&host_data, tasks);

    cudaStatus = hipMalloc((void**)&dev_data, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed for data array!";
        exit(1);
    }

    host_results = new long long[tasks.size()];
    cudaStatus = hipMalloc((void**)&dev_results, tasks.size() * sizeof(long long));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed for results array!";
        hipFree(dev_data);
        exit(1);
    }

    cudaStatus = hipMemcpy(dev_data, host_data, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed: data array to device!";
        hipFree(dev_results);
        hipFree(dev_data);
        exit(1);
    }


    hipEventRecord(start, 0);
    minmaxKernel << <gridSize, blockSize >> > (tasks.size(), dev_results, dev_data, size);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Kernel elapsed time:  %3.3f ms \n", time);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(dev_results);
        hipFree(dev_data);
        exit(1);
    }

    cudaStatus = hipMemcpy(host_results, dev_results, tasks.size() * sizeof(long long), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed: results array to host!";
        hipFree(dev_results);
        hipFree(dev_data);
        exit(1);
    }

    map<unsigned int, long long> results;
    auto it = tasks.begin();
    for (int i = 0; i < tasks.size(); i++, it++)
        results.insert(pair<unsigned int, long long>(it->GetTreePosition(), host_results[i]));

    Board bestMove = mmAlg.GetBestMove(board, player, results, searchedDepth);

    hipFree(dev_results);
    hipFree(dev_data);

    return bestMove;
}

int main()
{
    Board _board(3, 3, 3);
    int depth = _board.GetRows() * _board.GetColumns();

    _board.SetCell(0, 0, OPPONENT); // the user
    depth--;

    Board bMove = FindBestMove(_board, PLAYER, depth);

    _board = bMove;
    depth--;

    return 0;
}