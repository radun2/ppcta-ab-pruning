#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

// added by me
#include <iostream>
#include <queue>
#include <map>

#include "board.h"
#include "minmax.h"
#include <stdlib.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void c(int* c, int count) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    for (; i < count; i += blockDim.x * gridDim.x)
        c[i] = (1 << i) + 1;
}

int _main() {
	Board myBoard(3, 3, 3);
	int xpostion = 0;
	int ypostion = 0;

	while (1) {
		myBoard.Print();
		if (myBoard.IsTerminal()) {
			cout << "You something...";
			cin >> xpostion;
			break;
		}
		cout << endl << "Provide postion to set 'X' mark" << endl;
		
		//Validate position
		bool posIsTaken = true;
		//Check if pos is taken
		while (posIsTaken == true) {
			//init vars
			posIsTaken = false;
			xpostion = 0;
			ypostion = myBoard.GetRows() + 2;
			//Check if x is valid
			while (1) {
				cout << "x(1-" << myBoard.GetColumns() << "):"; cin >> xpostion;
				xpostion--;
				if (xpostion < 0 || xpostion >= myBoard.GetColumns()) {
					system("cmd /c cls");
					myBoard.Print();
					cout << endl << "Provide postion to set 'X' mark" << endl;
					cout << "Error: x value is not valid" << endl;
				}
				else {
					break;
				}
			}

			//Check if y is valid
			while (1) {
				cout << "y(1-" << myBoard.GetRows() << "):"; cin >> ypostion;
				ypostion--;
				if (ypostion < 0 || ypostion >= myBoard.GetRows()) {
					system("cmd /c cls");
					myBoard.Print();
					cout << endl << "Provide postion to set 'X' mark" << endl;
					cout << "Error: y value is not valid" << endl;
					cout << "x(1-" << myBoard.GetColumns() << "):" << xpostion << endl;
				}
				else {
					break;
				}
			}

			//Check pos is taken
			if (myBoard.GetCell(xpostion, ypostion) != 0) {
				posIsTaken = true;
				system("cmd /c cls");
				myBoard.Print();
				cout << endl << "Provide postion to set 'X' mark" << endl;
				cout << "Error: postion is taken" << endl;
			}
		}

		//Add pos and contiune
		myBoard.SetCell(xpostion, ypostion, PLAYER);
		//TO DO LOGIC
		//myBoard.SetCell(0, 0, OPPONENT);
		system("cmd /c cls");
	}

	return 0;
}

int main()
{
	return _main();
    int N = 2 << 13;
    Board _board(2, 2, 2);
    ppca::minmax mmAlg;

    GAME_CHAR player = PLAYER;

    int depth = _board.GetColumns() * _board.GetRows(),
        searchedDepth;

    int blockSize;      // The launch configurator returned block size 
    int minGridSize;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch 
    int gridSize;       // The actual grid size needed, based on input size

    auto tasks = mmAlg.GetTasks(_board, player, N, depth, searchedDepth);
    map<unsigned int, long long int> results;

    while (!tasks.empty()) {
        Board& b = tasks.front();
        b.CalculateScore();
        results.insert(pair<unsigned int, long long int>(b.GetTreePosition(), rand() - (RAND_MAX / 2)));

        tasks.pop();
    }
    Board nextMove;
    mmAlg.GetBestMove(_board, player, results, depth);

    return 0;
    tasks = mmAlg.GetTasks(_board, player, N, depth, searchedDepth);


    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, c, 0, tasks.size());

    // Round up according to array size 
    gridSize = (tasks.size() + blockSize - 1) / blockSize;

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Occupancy calculator elapsed time:  %3.3f ms \n", time);


    hipEventRecord(start, 0);
    c << <gridSize, blockSize >> > (nullptr, 0);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Kernel elapsed time:  %3.3f ms \n", time);

    auto cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }

    return 0;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return 1;
    }

    int size = 400000000;
    int* dev_c = 0;
    int* h_c = new int[size];
    memset(h_c, 0, sizeof(int)*size);

    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    cudaStatus = hipMemcpy(dev_c, h_c, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    c << <14, 1024 >> > (dev_c, size);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return 0;
    }

    cudaStatus = hipMemcpy(h_c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    hipFree(dev_c);
    return 0;

    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel << <1, size >> > (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}

